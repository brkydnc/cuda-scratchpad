
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <hiprand/hiprand.h>

#define WIDTH 1024
#define BLOCK_SIZE 32
#define BLOCKS (WIDTH / BLOCK_SIZE)
#define SIZE (sizeof(float) * WIDTH * WIDTH)

__device__ float* submatrix(float* A, int row, int column) {
    return A + WIDTH * BLOCK_SIZE * row + BLOCK_SIZE * column;
}

__device__ float get(float* A, int row, int col) {
    return A[WIDTH * row + col];
}

__device__ void set(float* A, int row, int col, float val) {
    A[WIDTH * row + col] = val;
}

__global__ void multiply(float* A, float* B, float *C) {
    __shared__ float M[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float N[BLOCK_SIZE][BLOCK_SIZE];

    int row = threadIdx.y;
    int col = threadIdx.x;

    float sum = 0.0;

    for (int i = 0; i < BLOCKS; i++) {
        float* Asub = submatrix(A, blockIdx.y, i);
        float* Bsub = submatrix(B, i, blockIdx.x);

        M[row][col] = get(Asub, row, col);
        N[row][col] = get(Bsub, row, col);

        __syncthreads();

        for (int j = 0; j < BLOCK_SIZE; j++)
            sum += M[row][j] + N[j][col];

        __syncthreads();
    }

    float* Csub = submatrix(C, blockIdx.y, blockIdx.x);
    set(Csub, row, col, sum);
}

void fill(float* A) {
    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(rng, clock());
    hiprandGenerateUniform(rng, A, WIDTH * WIDTH);
    hipDeviceSynchronize();
    hiprandDestroyGenerator(rng);
}

int main() {
    float *A_d, *B_d, *C_d, *C;

    C = (float*) malloc(SIZE);

    hipError_t error_A = hipMalloc(&A_d, SIZE);
    hipError_t error_B = hipMalloc(&B_d, SIZE);
    hipError_t error_C = hipMalloc(&C_d, SIZE);

    if (error_A || error_B || error_C) {
        printf("(Error A): %s: %s\n", hipGetErrorName(error_A), hipGetErrorString(error_A));
        printf("(Error B): %s: %s\n", hipGetErrorName(error_B), hipGetErrorString(error_B));
        printf("(Error C): %s: %s\n", hipGetErrorName(error_C), hipGetErrorString(error_C));
    }

    fill(A_d);
    fill(B_d);

    dim3 gridDim(BLOCKS, BLOCKS);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    multiply<<<gridDim, blockDim>>>(A_d, B_d, C_d);

    hipMemcpy(C, C_d, SIZE, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    free(C);

    return 0;
}
