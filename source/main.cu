
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

#define WIDTH 4
#define BLOCK_SIZE 2
#define BLOCKS (WIDTH / BLOCK_SIZE)
#define SIZE (sizeof(float) * WIDTH * WIDTH)

__device__ float* submatrix(float* A, int row, int column) {
    return A + WIDTH * BLOCK_SIZE * row + BLOCK_SIZE * column;
}

__device__ float get(float* A, int row, int col) {
    return A[WIDTH * row + col];
}

__device__ void set(float* A, int row, int col, float val) {
    A[WIDTH * row + col] = val;
}

__global__ void multiply(float* A, float* B, float *C) {
    __shared__ float M[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float N[BLOCK_SIZE][BLOCK_SIZE];

    int row = threadIdx.y;
    int col = threadIdx.x;

    float sum = 0.0;

    for (int i = 0; i < BLOCKS; i++) {
        float* Asub = submatrix(A, blockIdx.y, i);
        float* Bsub = submatrix(B, i, blockIdx.x);

        M[row][col] = get(Asub, row, col);
        N[row][col] = get(Bsub, row, col);

        __syncthreads();

        for (int j = 0; j < BLOCK_SIZE; j++)
            sum += M[row][j] + N[j][col];

        __syncthreads();
    }

    float* Csub = submatrix(C, blockIdx.y, blockIdx.x);
    set(Csub, row, col, sum);
}

void cublasMultiply(float* B, float *A, float *C) {
    const float alpha { 1.0f };
    const float beta { 0.0f };
    hipblasHandle_t handle;

    hipblasCreate(&handle);

    hipblasSgemm(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T,
        WIDTH, WIDTH, WIDTH,
        &alpha,
        A, WIDTH,
        B, WIDTH,
        &beta,
        C, WIDTH
    );

    hipblasDestroy(handle);
}

void fill(float* A) {
    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(rng, 123);
    hiprandGenerateUniform(rng, A, WIDTH * WIDTH);
    hipDeviceSynchronize();
    hiprandDestroyGenerator(rng);
}

void display(float *A) {
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%f,", A[WIDTH * i + j]);
        }
        printf("\n");
    }
}

int main() {
    float *A, *B, *C, *D;
    float *A_d, *B_d, *C_d, *D_d;

    A = (float*) malloc(SIZE);
    B = (float*) malloc(SIZE);
    C = (float*) malloc(SIZE);
    D = (float*) malloc(SIZE);

    hipError_t error_A = hipMalloc(&A_d, SIZE);
    hipError_t error_B = hipMalloc(&B_d, SIZE);
    hipError_t error_C = hipMalloc(&C_d, SIZE);
    hipError_t error_D = hipMalloc(&D_d, SIZE);

    if (error_A || error_B || error_C) {
        printf("(Error A): %s: %s\n", hipGetErrorName(error_A), hipGetErrorString(error_A));
        printf("(Error B): %s: %s\n", hipGetErrorName(error_B), hipGetErrorString(error_B));
        printf("(Error C): %s: %s\n", hipGetErrorName(error_C), hipGetErrorString(error_C));
        printf("(Error C): %s: %s\n", hipGetErrorName(error_D), hipGetErrorString(error_D));
    }

    fill(A_d);
    fill(B_d);

    hipMemcpy(A, A_d, SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(B, B_d, SIZE, hipMemcpyDeviceToHost);

    printf("Matrix A:\n");
    display(A);
    printf("\n");
    printf("Matrix B:\n");
    display(B);

    dim3 gridDim(BLOCKS, BLOCKS);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    multiply<<<gridDim, blockDim>>>(A_d, B_d, C_d);

    cublasMultiply(A_d, B_d, D_d);

    hipMemcpy(C, C_d, SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(D, D_d, SIZE, hipMemcpyDeviceToHost);

    printf("\n");
    printf("Matrix C (multiply):\n");
    display(C);
    printf("\n");
    printf("Matrix D (cuBLAS):\n");
    display(D);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipFree(D_d);

    free(A);
    free(B);
    free(C);
    free(D);

    return 0;
}
